#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

#define SUBBLOCK_COLS (BLOCKTILE_N / BLOCKDIM_X)
#define SUBBLOCKS_ROWS (BLOCKTILE_M / BLOCKDIM_Y)

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B)
{
    // if (threadIdx.x == 0 && threadIdx.y == 0) {
        // printf("blockDim.x = %d, blockDim.y = %d\n", blockDim.x, blockDim.y);
    // }
    // const int x = blockDim.x;
    // const int y = blockDim.y;
    __shared__ _DOUBLE_ shared_A[BLOCKTILE_M][BLOCKTILE_K];
    __shared__ _DOUBLE_ shared_B[BLOCKTILE_K][BLOCKTILE_N];

    int matrix_size = N * N;
    int ty = threadIdx.y, tx = threadIdx.x;
    int by = blockIdx.y, bx = blockIdx.x;
    int I = by * BLOCKTILE_M + ty;
    int J = bx * BLOCKTILE_N + tx;
    // if (threadIdx.x != 0 && threadIdx.y != 0) {
        // printf("I = %d, J = %d\n", I, J);
        // printf("bx = %d, by = %d\n", bx, by);
    // }
    _DOUBLE_ Cij[SUBBLOCKS_ROWS][SUBBLOCK_COLS] = {0};

    #pragma unroll
    for (int kk = 0; kk < N; kk += BLOCKTILE_K) {

        /* 
         * Requirements:
         *
         * 1) Increase ILP by doing more work per thread. This reduces
         * the number of threads needed per thread-block and therefore
         * reduces the  amount of time spent waiting in the 
         * __syncthreads() barrier. By using independent instructions 
         * before and after the __syncthreads() barrier, we keep the
         * GPU busy despite the lower occupancy resulting from smaller
         * thread block sizes.
         *
         * 2) Global memory accesses should coalesce i.e. threads within a
         * warp must access sequential memory addresses aligned on a 128-
         * byte boundary.
         *
         * We achieve the above two goals by using smaller thread block
         * sizes (around 16x16 as provided in the starter code). However,
         * since the matrix block dimensions could be larger than the thread
         * block sizes, we can improve ILP by getting each thread to load
         * more than one element of matrices A and B. For example, using
         * 64x64 matrix block sizes with a thread block of 16x16 would mean
         * that each thread is tasked with now loading 4x4 elements of 
         * matrices A and B. But while doing this, we need to ensure that
         * global memory accesses are coalesced. Therefore, we split the
         * matrix block into smaller sub-blocks such that each thread loads
         * an element from each of these sublocks (essentially, the thread
         * block size acts as a stride for each thread to load elements).
         * This way, threads with consecutive thread IDs would access
         * sequential global memory addresses.
         */
        #pragma unroll        
        for (int i = 0; i < BLOCKTILE_M; i += BLOCKDIM_Y) {
            #pragma unroll
            for (int j = 0; j < BLOCKTILE_K; j += BLOCKDIM_X) {

                int A_index = (I + i) * N + kk + tx + j;

                shared_A[ty + i][tx + j] = (A_index < matrix_size) ? A[A_index] : 0;
            }
        }

        for (int i = 0; i < BLOCKTILE_K; i += BLOCKDIM_Y) {
            #pragma unroll
            for (int j = 0; j < BLOCKTILE_N; j += BLOCKDIM_X) {
                int B_index = (kk + ty + i) * N + J + j;

                shared_B[ty + i][tx + j] = (B_index < matrix_size) ? B[B_index] : 0;
            }
        }

        __syncthreads();

        /*
         * Shared memory accesses should be such that they don't cause bank conflicts.
         * To do this, we once again use striding across matrix sub-blocks so that
         * threads with consecutive IDs access different banks.
         */

        #pragma unroll
        for (int k = 0; k < BLOCKTILE_K; ++k) {
            #pragma unroll
            for (int i = 0; i < SUBBLOCKS_ROWS; ++i) {
                #pragma unroll
                for (int j = 0; j < SUBBLOCK_COLS; ++j) {
                    Cij[i][j] += shared_A[ty + i * BLOCKDIM_Y][k] * shared_B[k][tx + j * BLOCKDIM_X];
                }
            }
        }

        __syncthreads();
    }

    #pragma unroll
    for (int i = 0; i < SUBBLOCKS_ROWS; ++i) {
        #pragma unroll
        for (int j = 0; j < SUBBLOCK_COLS; ++j) {
            if (I + i * BLOCKDIM_Y < N && J + j * BLOCKDIM_X < N) {
                C[(I + i * BLOCKDIM_Y) * N + J + j * BLOCKDIM_X] = Cij[i][j];
            }
        }
    }
}
